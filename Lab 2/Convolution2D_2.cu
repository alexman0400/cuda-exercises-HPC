#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005
 
void cudaCheckErrors() {
	hipError_t error = hipGetLastError();
    if(error != hipSuccess){
		// something's gone wrong
		// print out the CUDA error as a string
		printf("CUDA Error: %s\n", hipGetErrorString(error));
		// we can't recover from the error -- exit the program
		exit(0);
	}
}
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR, int check) {
	int k;
	float sum = 0;
	int x = threadIdx.x;
	int y = blockIdx.x;
	for (k = -filterR; k <= filterR; k++) {
		int d = x + k;

		if (d >= 0 && d < imageW) {
		  sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
		}     

		h_Dst[y * imageW + x] = sum;
	}
}

__global__ void convolutionColumnGPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR, int check) {
	int k;
	float sum = 0;
	int x = threadIdx.x;
	int y = blockIdx.x;
	for (k = -filterR; k <= filterR; k++) {
		int d = y + k;

		if (d >= 0 && d < imageH) {
		  sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
		}   

		h_Dst[y * imageW + x] = sum;
	  }
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
	struct timespec  tv1, tv2;
	
	float error,maxerror=0;
	
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
	*h_Output,
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
	elapsedTime;

	int imageW;
    int imageH;
    unsigned int i;
	
	dim3 threadsPerBlock, blocks;
	
	hipEvent_t S,E;
	hipEventCreate(&S);
	cudaCheckErrors();
	hipEventCreate(&E);
	cudaCheckErrors();
	
	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
	h_Output = (float *)malloc(imageW * imageH * sizeof(float));
    if (h_Filter==NULL || h_Input==NULL || h_Buffer==NULL || h_OutputCPU==NULL) {
      printf("error with malloc");
      return 0;
    }

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }


    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
	
	
			
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
	printf("Allocating and initializing device arrays...\n");
    hipMalloc( (void**) &d_Filter, FILTER_LENGTH * sizeof(float));
	cudaCheckErrors();
    hipMalloc( (void**) &d_Input, imageW * imageH * sizeof(float));
	cudaCheckErrors();
    hipMalloc( (void**) &d_Buffer, imageW * imageH * sizeof(float));
	cudaCheckErrors();
    hipMalloc( (void**) &d_OutputGPU, imageW * imageH * sizeof(float));
	cudaCheckErrors();
    
	hipMemset(d_Buffer, 0, imageW * imageH);
	cudaCheckErrors();
	hipMemset(d_OutputGPU, 0, imageW * imageH);
    cudaCheckErrors();
	
	hipEventRecord(S, NULL);
	cudaCheckErrors();
	
    printf("Copying the arrays from host to device...\n");
    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	cudaCheckErrors();
	
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	cudaCheckErrors();
    
    printf("GPU computation...\n");
	
	threadsPerBlock.x = imageH;
	blocks.x = imageH;
	threadsPerBlock.y=1;
	blocks.y=1;
	
	printf("Number of blocks = %d * %d\nNumber of threads per block = %d * %d\n",blocks.x, blocks.y, threadsPerBlock.x, threadsPerBlock.y);
	convolutionRowGPU<<<blocks,threadsPerBlock>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius, 1); // convolution kata grammes
	hipDeviceSynchronize();
	cudaCheckErrors();
	
	convolutionColumnGPU<<<blocks,threadsPerBlock>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius, 1); // convolution kata sthles
	hipDeviceSynchronize();
	cudaCheckErrors();
	
	hipMemcpy(h_Output, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);
	cudaCheckErrors();
    
	hipEventRecord(E, NULL);
	cudaCheckErrors();
	hipEventSynchronize(E);
	cudaCheckErrors();
	
	hipEventElapsedTime(&elapsedTime, S, E); 
	cudaCheckErrors();
	
	printf ("CPU execution time = %10g seconds\n",
			(float) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(float) (tv2.tv_sec - tv1.tv_sec));
	printf("GPU execution Time: %f\n", elapsedTime); // Print Elapsed time
	
	// Destroy CUDA Event API Events
	hipEventDestroy(S);
	cudaCheckErrors();
	hipEventDestroy(E);
	cudaCheckErrors();
	
	//error calculation
    for (i = 0; i < imageW * imageH; i++) {
		error = ABS(h_Output[i] - h_OutputCPU[i]);
		if(error>accuracy){
			maxerror = error;
			//printf("error spotted\n");
			// free all the allocated memory
			// free(h_OutputCPU);
			// free(h_Buffer);
			// free(h_Input);
			// free(h_Filter);
			// free(h_Output);
			// hipFree(d_OutputGPU);
			// hipFree(d_Buffer);
			// hipFree(d_Input);
			// hipFree(d_Filter);
			
			// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
			// hipDeviceReset();
			// return 0;
		}
	}
	printf("maxerror is: %f\n", maxerror);
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_Output);
	hipFree(d_OutputGPU);
	hipFree(d_Buffer);
    hipFree(d_Input);
    hipFree(d_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
