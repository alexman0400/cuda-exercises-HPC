



//o hist na paei sthn texture memory
//cdf += paromoio paradeigma sto pdf




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define BANKNUM 32

typedef struct{
    int w;
    int h;
    unsigned char * img;
} PGM_IMG;    

void run_cpu_gray_test(PGM_IMG img_in, char *out_filename);
void run_gpu_gray_test(PGM_IMG img_in, char *out_filename);
__global__ void histogram_kernel(unsigned char * img_in, 
                            int * hist_in, int img_size, int min, int d);
void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin);
void histogram_equalization(unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin);
PGM_IMG contrast_enhancement_g(PGM_IMG img_in);
void free_pgm(PGM_IMG img);
void write_pgm(PGM_IMG img, const char * path);
PGM_IMG read_pgm(const char * path);

void cudaCheckErrors() {
	hipError_t error = hipGetLastError();
    if(error != hipSuccess){
		// something's gone wrong
		// print out the CUDA error as a string
		printf("CUDA Error: %s\n", hipGetErrorString(error));
		// we can't recover from the error -- exit the program
		exit(0);
	}
}

int main(int argc, char *argv[]){
    PGM_IMG img_ibuf_g;
	
	if (argc != 3) {
		printf("Run with input file name and output file name as arguments\n");
		exit(1);
	}
	
    printf("Running contrast enhancement for gray-scale images.\n");
    img_ibuf_g = read_pgm(argv[1]);
    run_cpu_gray_test(img_ibuf_g, argv[2]);
	run_gpu_gray_test(img_ibuf_g, argv[2]);
    free_pgm(img_ibuf_g);

	return 0;
}

void run_gpu_gray_test(PGM_IMG img_in, char *out_filename)
{
	int hist[256], i = 0, min = 0, d, *d_Hist;
	float elapsedTime;
	unsigned char *d_Img;
	
	hipEvent_t S,E;
	
	hipEventCreate(&S);
	cudaCheckErrors();
	
	hipEventCreate(&E);
	cudaCheckErrors();
	
	PGM_IMG	h_Result;
	
	hipStream_t stream1;
	cudaCheckErrors();
	hipStreamCreate(&stream1);
	cudaCheckErrors();
	
	hipHostMalloc((void**)&h_Result.img, sizeof(unsigned char)*(img_in.h * img_in.w), hipHostMallocDefault);
	
	h_Result.w = img_in.w;
	h_Result.h = img_in.h;
	
	hipEventRecord(S, NULL);
	cudaCheckErrors();
	
	
	
	//Memory allocation on GPU
	printf("Allocating GPU memory...\n");
	hipMalloc( (void**) &d_Img, img_in.w * img_in.h * sizeof(unsigned char));
	cudaCheckErrors();
	hipMemcpyAsync(d_Img, img_in.img, img_in.w * img_in.h * sizeof(unsigned char), hipMemcpyHostToDevice, stream1);
	cudaCheckErrors();
	
	histogram(hist, img_in.img, img_in.h * img_in.w, 256);
	
	hipMalloc( (void**) &d_Hist, 256 * sizeof(int));
	cudaCheckErrors();
	hipMemcpyAsync(d_Hist, hist, 256 * sizeof(256), hipMemcpyHostToDevice, stream1);
	cudaCheckErrors();
	
	printf("Starting GPU processing...\n");
	
	//Kernel
	while(min == 0){
        min = hist[i++];
    }
	d = img_in.w * img_in.h - min;
	histogram_kernel<<<img_in.h*img_in.w/1024, 1024, 0, stream1>>>(d_Img,d_Hist, img_in.w * img_in.h, min, d);
    
	hipMemcpyAsync(h_Result.img, d_Img, img_in.w * img_in.h * sizeof(unsigned char), hipMemcpyDeviceToHost, stream1);
	cudaCheckErrors();
	
	hipEventRecord(E, NULL);
	cudaCheckErrors();
	hipEventSynchronize(E);
	cudaCheckErrors();
	
	hipEventElapsedTime(&elapsedTime, S, E); 
	cudaCheckErrors();
	
	printf("GPU execution Time: %f\n", elapsedTime); // Print Elapsed time
	
	hipDeviceSynchronize();
    write_pgm(h_Result, out_filename);
	
	// Destroy CUDA Event API Events
	hipEventDestroy(S);
	cudaCheckErrors();
	hipEventDestroy(E);
	cudaCheckErrors();
	
	//destroy stream
	hipStreamDestroy(stream1);
	cudaCheckErrors();
	
	//free memory everywhere
	hipFree(d_Img);
	cudaCheckErrors();
	hipFree(d_Hist);
	cudaCheckErrors();
    hipHostFree(h_Result.img);
	cudaCheckErrors();
	
	hipDeviceReset();
	cudaCheckErrors();
}

__global__ void histogram_kernel(unsigned char * img_in, 
                            int * hist_in, int img_size, int min, int d) {
	int pos = threadIdx.x + blockIdx.x*blockDim.x;
	__shared__ int lut[256];
	__shared__ int cdf[256];
	
	if (threadIdx.x < 256) {
		cdf[threadIdx.x] = hist_in[threadIdx.x];
		////////__device__
		
		for (int offset = 1; offset < 256; offset *= 2) {
			__syncthreads();
			if (threadIdx.x >= offset)
				cdf[threadIdx.x] += cdf[threadIdx.x - offset];
		}
		lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
        if(lut[threadIdx.x] < 0){
            lut[threadIdx.x] = 0;
        }
	}
	__syncthreads();
	if(lut[img_in[pos]] > 255){
		img_in[pos] = 255;
	}
	else{
		img_in[pos] = (unsigned char)lut[img_in[pos]];
	}
}

void run_cpu_gray_test(PGM_IMG img_in, char *out_filename)
{
    PGM_IMG img_obuf;
    
    
    printf("Starting CPU processing...\n");
    img_obuf = contrast_enhancement_g(img_in);
    write_pgm(img_obuf, out_filename);
    free_pgm(img_obuf);
}


PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256];
    
    
    PGM_IMG result;
    int v_max;//, i;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }
    
    fscanf(in_file, "%s", sbuf); /*Skip the magic number*/
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);
    printf("Image size: %d x %d\n", result.w, result.h);
    

    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

        
    fread(result.img,sizeof(unsigned char), result.w*result.h, in_file);    
    fclose(in_file);
    
    return result;
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    free(img.img);
}

void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    int i;
    for ( i = 0; i < nbr_bin; i ++){
        hist_out[i] = 0;
    }

    for ( i = 0; i < img_size; i ++){
        hist_out[img_in[i]] ++;
    }
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin){
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++];
    }
    d = img_size - min;
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }
        
        
    }
    
    /* Get the result image */
    for(i = 0; i < img_size; i ++){
        if(lut[img_in[i]] > 255){
            img_out[i] = 255;
        }
        else{
            img_out[i] = (unsigned char)lut[img_in[i]];
        }
        
    }
}

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result;
    int hist[256];
    
    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);
    return result;
}
